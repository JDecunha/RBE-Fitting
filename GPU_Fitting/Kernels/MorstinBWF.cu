
#include <hip/hip_runtime.h>
__global__ void MorstinBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
	    double BWFVal = (BWFParams[0]*((1-exp(-binCenter[i]*BWFParams[1]-BWFParams[2]*binCenter[i]*binCenter[i]-BWFParams[3]*binCenter[i]*binCenter[i]*binCenter[i]))/binCenter[i]));

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

