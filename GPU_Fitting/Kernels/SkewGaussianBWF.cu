
#include <hip/hip_runtime.h>
__global__ void SkewGaussianBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double mag = BWFParams[0];
		double xi = BWFParams[1];
	    double omega = BWFParams[2];
	    double alpha = BWFParams[3];
	    double arg = (binCenter[i] - xi) / omega;

	    //Just the gaussian function
	    double smallphi = BWFParams[0]*exp(-((binCenter[i]-BWFParams[1])*(binCenter[i]-BWFParams[1]))/(BWFParams[2]*BWFParams[2]*2));

	    //This should work because there is a cuda version of erf actually
	    double bigphi = (1 + erf(alpha * arg/sqrt(2.)));

	    //Put it all together
	    double BWFVal = (smallphi * bigphi);

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

