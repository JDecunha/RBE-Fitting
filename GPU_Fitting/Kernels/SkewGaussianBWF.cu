#include "hip/hip_runtime.h"
__global__ void SkewGaussianBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{

		double xi = BWFParams[1];
	    double omega = BWFParams[2];
	    double alpha = BWFParams[3];
	    double arg = (binCenter[i] - xi) / omega;
	    double smallphi = Gaus(arg, 0.0, 1.0, true);
	    double bigphi = 0.5 * (1 + erf(alpha * arg/sqrt(2)));
	    double BWFVal = ((2./omega) * smallphi * bigphi)+BWFParams[0];

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

