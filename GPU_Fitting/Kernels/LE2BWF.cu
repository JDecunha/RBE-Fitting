
#include <hip/hip_runtime.h>
__global__ void LE2BWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double BWFVal = (BWFParams[1]*binCenter[i])+exp(-BWFParams[2]*binCenter[i]*binCenter[i])+BWFParams[0];

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

