
#include <hip/hip_runtime.h>
__global__ void GaussianBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{

		double BWFVal = BWFParams[1]*exp(-((binCenter[i]-BWFParams[2])*(binCenter[i]-BWFParams[2]))/(BWFParams[3]*BWFParams[3]*2))+BWFParams[0];

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

