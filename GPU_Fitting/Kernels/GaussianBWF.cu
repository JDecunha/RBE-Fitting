
#include <hip/hip_runtime.h>
__global__ void GaussianBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{

		double BWFVal = BWFParams[0]*exp(-((binCenter[i]-BWFParams[1])*(binCenter[i]-BWFParams[1]))/(BWFParams[2]*BWFParams[2]*2));

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

