
#include <hip/hip_runtime.h>
__global__ void LEBWFPenaltyFunction(const double* binWidth, const double* binCenter,  const int* numBins, const double * BWFParams, double* negativeArea, double* totalArea)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double BWFVal = (BWFParams[1]*binCenter[i])+exp(-BWFParams[2]*binCenter[i])+BWFParams[0];

		double area = binWidth[i]*BWFVal;

		if (area < 0)
		{
			area = -area;
			negativeArea[i] = area;
			totalArea[i] = area;
		}
		else
		{
			negativeArea[i] = 0;
			totalArea[i] = area;
		}
	}
}

