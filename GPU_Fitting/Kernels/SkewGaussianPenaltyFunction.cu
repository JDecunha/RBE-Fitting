#include "hip/hip_runtime.h"
__global__ void SkewGaussianPenaltyFunction(const double* binWidth, const double* binCenter,  const int* numBins, const double * BWFParams, double* negativeArea, double* totalArea)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double xi = BWFParams[1];
	    double omega = BWFParams[2];
	    double alpha = BWFParams[3];
	    double arg = (binCenter[i] - xi) / omega;
	    double smallphi = Gaus(arg, 0.0, 1.0, true);
	    double bigphi = 0.5 * (1 + erf(alpha * arg/sqrt(2)));
	    double BWFVal = ((2./omega) * smallphi * bigphi)+BWFParams[0];

		double area = binWidth[i]*BWFVal;

		if (area < 0)
		{
			area = -area;
			negativeArea[i] = area;
			totalArea[i] = area;
		}
		else
		{
			negativeArea[i] = 0;
			totalArea[i] = area;
		}
	}
}

