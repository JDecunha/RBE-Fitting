
#include <hip/hip_runtime.h>
__global__ void SkewGaussianPenaltyFunction(const double* binWidth, const double* binCenter,  const int* numBins, const double * BWFParams, double* negativeArea, double* totalArea)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double mag = BWFParams[1];
		double xi = BWFParams[2];
	    double omega = BWFParams[3];
	    double alpha = BWFParams[4];
	    double arg = (binCenter[i] - xi) / omega;

	    //Just the gaussian function
	    double smallphi = BWFParams[1]*exp(-((binCenter[i]-BWFParams[2])*(binCenter[i]-BWFParams[2]))/(BWFParams[3]*BWFParams[3]*2));

	    //This should work because there is a cuda version of erf actually
	    double bigphi = 0.5 * (1 + erf(alpha * arg/sqrt(2.)));

	    //Put it all together
	    double BWFVal = ((2./omega) * smallphi * bigphi)+BWFParams[0];

		double area = binWidth[i]*BWFVal;

		if (area < 0)
		{
			area = -area;
			negativeArea[i] = area;
			totalArea[i] = area;
		}
		else
		{
			negativeArea[i] = 0;
			totalArea[i] = area;
		}
	}
}

