
#include <hip/hip_runtime.h>
__global__ void SkewGaussianPenaltyFunction(const double* binWidth, const double* binCenter,  const int* numBins, const double * BWFParams, double* negativeArea, double* totalArea)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double mag = BWFParams[0];
		double xi = BWFParams[1];
	    double omega = BWFParams[2];
	    double alpha = BWFParams[3];
	    double arg = (binCenter[i] - xi) / omega;

	    //Just the gaussian function
	    double smallphi = BWFParams[0]*exp(-((binCenter[i]-BWFParams[1])*(binCenter[i]-BWFParams[1]))/(BWFParams[2]*BWFParams[2]*2));

	    //This should work because there is a cuda version of erf actually
	    double bigphi = (1 + erf(alpha * arg/sqrt(2.)));

	    //Put it all together
	    double BWFVal = (smallphi * bigphi);

		double area = binWidth[i]*BWFVal;

		if (area < 0)
		{
			area = -area;
			negativeArea[i] = area;
			totalArea[i] = area;
		}
		else
		{
			negativeArea[i] = 0;
			totalArea[i] = area;
		}
	}
}

