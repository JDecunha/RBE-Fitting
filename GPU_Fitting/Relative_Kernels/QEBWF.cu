
#include <hip/hip_runtime.h>
__global__ void QEBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double BWFVal = (BWFParams[1]*binCenter[i]*binCenter[i])+exp(-BWFParams[2]*binCenter[i])+BWFParams[0];

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

