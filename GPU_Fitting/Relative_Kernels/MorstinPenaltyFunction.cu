
#include <hip/hip_runtime.h>
__global__ void MorstinPenaltyFunction(const double* binWidth, const double* binCenter,  const int* numBins, const double * BWFParams, double* negativeArea, double* totalArea)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double BWFVal = (BWFParams[0]*((1-exp(-binCenter[i]*BWFParams[1]-BWFParams[2]*binCenter[i]*binCenter[i]-BWFParams[3]*binCenter[i]*binCenter[i]*binCenter[i]))/binCenter[i]))+BWFParams[4];

		double area = binWidth[i]*BWFVal;

		if (area < 0)
		{
			area = -area;
			negativeArea[i] = area;
			totalArea[i] = area;
		}
		else
		{
			negativeArea[i] = 0;
			totalArea[i] = area;
		}
	}
}

