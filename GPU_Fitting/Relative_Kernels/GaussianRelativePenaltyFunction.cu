
#include <hip/hip_runtime.h>
__global__ void GaussianPenaltyFunction(const double* binWidth, const double* binCenter,  const int* numBins, const double * BWFParams, double* negativeArea, double* totalArea)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double BWFVal = (BWFParams[0]*exp(-((binCenter[i]-BWFParams[1])*(binCenter[i]-BWFParams[1]))/(BWFParams[2]*BWFParams[2]*2)))+1;

		double area = binWidth[i]*BWFVal;

		if (area < 0)
		{
			area = -area;
			negativeArea[i] = area;
			totalArea[i] = area;
		}
		else
		{
			negativeArea[i] = 0;
			totalArea[i] = area;
		}
	}
}

