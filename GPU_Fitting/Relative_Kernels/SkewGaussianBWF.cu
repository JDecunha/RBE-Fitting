
#include <hip/hip_runtime.h>
__global__ void SkewGaussianBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		//double mag = BWFParams[1];
		double xi = BWFParams[2];
	    double omega = BWFParams[3];
	    double alpha = BWFParams[4];
	    double arg = (binCenter[i] - xi) / omega;

	    //Just the gaussian function
	    double smallphi = BWFParams[1]*exp(-((binCenter[i]-BWFParams[2])*(binCenter[i]-BWFParams[2]))/(BWFParams[3]*BWFParams[3]*2));

	    //This should work because there is a cuda version of erf actually
	    double bigphi = (1 + erf(alpha * arg/sqrt(2.)));

	    //Put it all together
	    double BWFVal = (smallphi * bigphi)+BWFParams[0];

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

