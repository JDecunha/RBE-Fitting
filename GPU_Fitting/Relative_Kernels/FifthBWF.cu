
#include <hip/hip_runtime.h>
__global__ void FifthBWF(const double* binWidth, const double* binCenter, const double* binValues, const int* numBins, const double * BWFParams, double* alphaList)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *numBins; i += blockDim.x * gridDim.x) 
	{
		double BWFVal = (binCenter[i]*binCenter[i]*binCenter[i]*binCenter[i]*binCenter[i]*BWFParams[5])+(binCenter[i]*binCenter[i]*binCenter[i]*binCenter[i]*BWFParams[4])+(binCenter[i]*binCenter[i]*binCenter[i]*BWFParams[3])+(binCenter[i]*binCenter[i]*BWFParams[2])+(binCenter[i]*BWFParams[1])+(BWFParams[0]);

		//Calculate the value of this bin (Integration is achieved by parallel summing later)
		alphaList[i] = binWidth[i]*binValues[i]*BWFVal;
	}
}

